/*
 * A CUDA solution for quadrature
 * integral = sum of areas trapezoidals that approximate curve
 * area of trapezoidal = mean height * width
 * 
 * (c) michael k bane
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>  // for wallclock timing functions
#include <stdio.h>
double get_wtime () {
/*
 * function to return double representing number of wall clock
 * seconds since some arbitrary point in time
 * mkbane (2023)
 * TO DO: update to clock_gettime 
 */
  struct timeval wallTime;
  gettimeofday(&wallTime, NULL);
  double seconds = wallTime.tv_sec;
  double microsecs = wallTime.tv_usec;
  return seconds + microsecs*1.0E-06;
}



// since we using static shared mem
#define maxThreads 1024

__global__
void calcMyArea(double x, double width, int numQuads, double *deviceBlockArea)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x; // thread num out of all threads across all blocks
  int p = threadIdx.x;                         // thread num on my given block
  //  int numBlocks = gridDim.x;
  int numThreads = blockDim.x;
  
  double myX;      // thread specific value of 'x'
  double fx, fxpw; // thread specific values of f(x) and f(x+width), respectively for given myX
  double height;        // mean height of rectangle
  
  if (numThreads > maxThreads) printf("exceeding static memory!\n");
  if (i < numQuads) {             // so if we have excess threads they do nothing
    __shared__ double myThreadArea[maxThreads]; // thread specific sub-area under curve
  
    // thread 0 per block to output info
    if (threadIdx.x == 0) {
      //DEBUG: printf("block %d has %d threads (cf %d quads)\n", blockIdx.x, numThreads, numQuads);
    }
    myThreadArea[p] = 0.0; // we accumulate area for given thread in this 'p' element of a shared area


    myX = x + i * width;
    // calc end points of rectangle and thus mean height
    // we hardwire the functions!
    fx = 20.4 + pow(myX,1.2)/3.1 - myX/log(3.0);
    fxpw = 20.4 + pow(myX+width,1.2)/3.1 - (myX+width)/log(3.0);
    height = 0.5*(fx+fxpw);
    myThreadArea[p] += height * width;
    //DEBUG: printf("block %d, thread %d with myX=%f now = %f\n", blockIdx.x, p, myX, myThreadArea[p]);

    // each thread now has a subarea, which we 'reduce' to give a block subarea saved in GPU shared memory
    // NB there are more efficient techniques available
    __syncthreads();
    if (p==0) {
      double myBlockArea = myThreadArea[0];
      // note that last block may not use all threads
      int myNumThreads = numThreads;
      if (blockIdx.x == gridDim.x - 1) {
	//DEBUG: printf("LAST BLOCK (#%d of %d) setting num threads to ", blockIdx.x, gridDim.x);
	myNumThreads = numQuads - blockIdx.x * blockDim.x;
	//DEBUG: printf("%d in block %d\n", myNumThreads, blockIdx.x);
      }
      //DEBUG: printf("block %d using %d threads in reduction step\n", blockIdx.x, myNumThreads);
      for (int t=1; t<myNumThreads; t++) {
	//DEBUG: printf("%f + %f = ", myBlockArea, myThreadArea[t]);
	myBlockArea += myThreadArea[t];
	//DEBUG: printf("%f\n", myBlockArea);
      }	
      // update GPU's global memory but only for this block's element
      //DEBUG: printf("blk updating element %d with value %f\n", blockIdx.x,myBlockArea);
      deviceBlockArea[blockIdx.x] = myBlockArea;
    }
  }
  else  {
    //DEBUG: printf("thread %d does nothing \n", i);
  }
}

int main(int argc, char *argv[]) {
  int numGPUs;
  double a,b; // bounds (user input)
  
  
  int numberQuads, blks, tpb;
  
  // parse input args
  if (argc==5) {
    a = atof(argv[1]);
    b = atof(argv[2]);
    numberQuads = atoi(argv[3]);
    tpb = atoi(argv[4]);
  }
  else {
    printf("need to enter:\n %s a b numQuads threadsPerBlock\n", argv[0]);
    abort();
  }

  printf("Integrating from %f to %f using %d quads and %d threads per block.\nHold tight!\n", a,b,numberQuads,tpb);

  
  // sync device for reliable timing
  hipDeviceSynchronize();
  double start = get_wtime();

  blks = (numberQuads + tpb - 1) / tpb;
  printf("using %d quads with %d blocks each using %d threads (total num threads: %d\n",
	 numberQuads, blks, tpb, blks*tpb);
  	
  const double width = (b-a) / (float) numberQuads;


  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  if (numGPUs >= 1 ) {
    printf("hello on CPU\n");
    /* call GPU kernel using b blocks and tpb threads per block  
     *
     * warning: the reduction only works for 1 block example
     * warning: do not amend value of 'b'
     *
     */

    // create variable array on device
    double *deviceBlockArea;
    hipMalloc(&deviceBlockArea, blks * sizeof(double));

    hipEvent_t startCuda, stopCuda;
    hipEventCreate(&startCuda); hipEventCreate(&stopCuda); // set them up, but not actually do timing
    hipEventRecord(startCuda,0);

    calcMyArea<<<blks, tpb>>> (a, width, numberQuads, deviceBlockArea);

    hipEventRecord(stopCuda,0);


    // check for device errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("CUDA RT error (post kernel) %s\n", hipGetErrorString(err));

    // now get results
    double integrand;
    integrand = 0.0;
    //    double localSum[blks];
    double *localSum;
    localSum = (double *) malloc(blks * sizeof(double));

    hipMemcpy(localSum, deviceBlockArea, blks * sizeof(double), hipMemcpyDeviceToHost);
    for (int i=0; i<blks; i++) {
      //DEBUG: printf("adding %f to integrand\n", localSum[i]);
      integrand += localSum[i];
    }	

    // check for device errors
    err = hipGetLastError();
    if (err != hipSuccess) printf("CUDA RT error (post memcpy) %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    double finish = get_wtime();
    printf("GPU [naive CUDA] integral: %f\n", integrand);
    // determine elapsed time of kernel only
    float eTime;
    hipEventElapsedTime(&eTime, startCuda, stopCuda);
    printf("%d tpb: WALL CLOCK Time: %f seconds (kernel: %f seconds)\n",tpb,(finish-start), eTime/1000.0);


  }

  else {
    printf("no GPU present\n");
  }

}

