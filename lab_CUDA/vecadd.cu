/* (c) mkbane@highendcompute.co.uk
   example of 1D grid to form A=B+C elemenwise
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


__global__ void vecadd(float *B, float *C, float *A, int N)
{
  // form A = B + C
  int idx=blockIdx.x*blockDim.x+threadIdx.x;

  if (idx<N) {
    A[idx] = B[idx] + C[idx];
  }

}

#include <stddef.h>
#include <sys/time.h>
double get_wtime () {
/*
 * function to return double representing number of wall clock
 * seconds since some arbitrary point in time
 * mkbane (2023)
 * TO DO: update to clock_gettime 
 */
  struct timeval wallTime;
  gettimeofday(&wallTime, NULL);
  double seconds = wallTime.tv_sec;
  double microsecs = wallTime.tv_usec;
  return seconds + microsecs*1.0E-06;
}

void serial_vecadd(float *B, float *C, float *A, int N)
{
  // form A = B + C
  int idx;

  for (idx=0; idx<N; idx++) {
    A[idx] = B[idx] + C[idx];
  }

}

int main(int argc, char *argv[]) {
  int i, N;
  float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    if (argc<=1) {
      printf("%s requires 'N'\n", argv[0]);
      return -1;
    }
    N = atoi(argv[1]);

    // device timer
    hipEvent_t startCuda, stopCuda;
    hipEventCreate(&startCuda); hipEventCreate(&stopCuda); // set them up, but not actually do timing

    int dSize = N*sizeof(float);
    hipMalloc(&d_A, dSize);
    hipMalloc(&d_B, dSize);
    hipMalloc(&d_C, dSize);
    A = (float *) malloc(dSize);
    B = (float *) malloc(dSize);
    C = (float *) malloc(dSize);

    for (i=0; i<N; i++) {
      B[i] = 100.0*(rand() - rand())/RAND_MAX;  
      C[i] = 0.1 + rand()/RAND_MAX;
    }

    hipMemcpy(d_B, B, dSize, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, dSize, hipMemcpyHostToDevice);
    // see if any errors
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess) {
      printf("(1) CUDA RT error: %s \n",hipGetErrorString(err));
    }

   // check (e.g.) 'deviceQuery' for device config maximums...

   int threadsPerBlock=32;
   int blocks = ceil((float)N/(float)threadsPerBlock);

   printf("For size %d, calling with %d blocks each of %d threads\n", N, blocks, threadsPerBlock);
   printf("1D grid => total of %d blocks, and total of %d threads\n",blocks,blocks*threadsPerBlock);

   hipEventRecord(startCuda,0);
   vecadd <<<blocks, threadsPerBlock>>> (d_B, d_C, d_A, N);
   hipEventRecord(stopCuda,0);

   // see if any errors launching/running kernel
   err = hipGetLastError();
   if ( err != hipSuccess) {
      printf("(2) CUDA RT error: %s \n",hipGetErrorString(err));
   }


    hipMemcpy(A, d_A, dSize, hipMemcpyDeviceToHost);
    // see if any errors
    err = hipGetLastError();
    if ( err != hipSuccess) {
      printf("(3) CUDA RT error: %s \n",hipGetErrorString(err));
    }


    // sample res so compiler not opt it all away
    i=N/2; 
    float eTime;
    hipEventElapsedTime(&eTime, startCuda, stopCuda);
    printf("GPU: A[%d] = %f (cf %f) in %f milliseconds\n", i, A[i], B[i]+C[i], eTime);
    
    
    /* run on CPU */
    double start, finish;
    start = get_wtime();
    serial_vecadd (B, C, A, N);
    finish = get_wtime();
    printf("CPU: A[%d] = %f (cf %f) in %f milliseconds\n", i, A[i], B[i]+C[i], 1000.*(finish-start));

}
